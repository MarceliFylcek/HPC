#include "hip/hip_runtime.h"
#include "utility.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <math.h>


__host__
void errorexit(const char *s) {
    printf("\n%s",s);	
    exit(EXIT_FAILURE);	 	
}


__global__
void calculate(int* result, int end, int size, long number) {
  int myindex = blockIdx.x*blockDim.x+threadIdx.x; 
   //each process
  int mine = myindex +2;
  int step = size;
  
  result[myindex] = 0;

  if (number > mine)
  {
    if(number%(long)mine == 0 )
    result[myindex] = 1;
  }
    
}

int main(int argc,char **argv) {

  Args ins__args;
  parseArgs(&ins__args, &argc, argv);
  
  //program input argument
  long inputArgument = ins__args.arg; 

  struct timeval ins__tstart, ins__tstop;
  gettimeofday(&ins__tstart, NULL);

  int threadsinblock = 1024;
  int blocksingrid = 10000;

  int size = threadsinblock * blocksingrid;
  int end=sqrt(inputArgument);
  int *dresults=NULL;
  int result = 0;


  if(size < inputArgument)
  {
    errorexit("Number too big");
  }

  int *hresults=(int*)malloc(size*sizeof(int));
  if (!hresults) errorexit("Error allocating memory on the host");


  // run your CUDA kernel(s) here

  //memory allocation on device (GPU)
  if (hipSuccess!=hipMalloc((void **)&dresults,size*sizeof(int)))
    errorexit("Error allocating memory on the GPU");

  //call kernel on GPU – calculation are executed on GPU
  calculate<<<blocksingrid,threadsinblock>>>(dresults, end, size, inputArgument);
  if (hipSuccess!=hipGetLastError())
    errorexit("Error during kernel launch");

  //copy all elements from device to host
  if (hipSuccess!=hipMemcpy(hresults,dresults,size*sizeof(int),hipMemcpyDeviceToHost))
      errorexit("Error copying results");


  // synchronize/finalize your CUDA computations
    for(int i=0;i<size;i++) {
      if (hresults[i]==1)
      {
        result = 1;
        break;
      }
    }

  gettimeofday(&ins__tstop, NULL);
  ins__printtime(&ins__tstart, &ins__tstop, ins__args.marker);

  if(result == 0)
  {
    printf("%ld is prime\n", inputArgument);
  }
  else
  {
      printf("%ld is not prime\n", inputArgument);
  }

}
